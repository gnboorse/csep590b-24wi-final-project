#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got HIP error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


__global__ void normalizerKernel() {
  
}

__global__ void sdtwKernel() {
  
}

int main(int argc, char **argv) {
 

  return 0;
}
